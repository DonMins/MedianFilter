#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h> 
#include ""
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "bmp/EasyBMP.h"
#include <stdio.h>
#include <time.h>

using namespace std;

#define WINDOW_SIZE 3
#define COUNT_POINTS 9

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
system("pause");\
}       


// объявляем ссылку на текстуру для двумерной текстуры float
texture<float, hipTextureType2D, hipReadModeElementType> tex;


__global__ void medianFilter(float *output, int imageWidth, int imageHeight) {
	//  выбрали строку и столбец для потока
	int col = blockIdx.x *  blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	// маска по которой будет находиться медиана
	float mask[COUNT_POINTS] = { 0,0,0,0,0,0,0,0,0 };
	
	int k = 0;
	// Т.к текстуры обладают свойством свертывание - т.е выход за границы, будем идти от -1 до 1 с шагом 1 по картинки, 
	//заполняя маску  
	for (int i = -1; i <= 1; i++) {
		for (int j = -1; j <= 1; j++) {
			mask[k] = tex2D(tex, col + j, row + i);
			k++;
		}
	}

	// Отсортировали значения в маске 
	for (int i = 1; i < COUNT_POINTS; i++) {
		for (int j = i; j > 0 && mask[j - 1] > mask[j]; j--) {
			int tmp = mask[j - 1];
			mask[j - 1] = mask[j];
			mask[j] = tmp;
		}
	}
	// в результирующий центральный пиксель записали медиану
	output[row * imageWidth + col] = mask[4];
	
}


float *readImage(char *filePathInput, unsigned int *rows, unsigned int *cols) {
	BMP Image;
	Image.ReadFromFile(filePathInput);
	*rows = Image.TellHeight();
	*cols = Image.TellWidth();
	float *imageAsArray = (float *)calloc(*rows * *cols, sizeof(float));
	// Преобразуем картику в черно-белую
	for (int i = 0; i < Image.TellWidth(); i++)	{
		for (int j = 0; j < Image.TellHeight(); j++) {
			double Temp = 0.30*(Image(i, j)->Red) +	0.59*(Image(i, j)->Green) +	0.11*(Image(i, j)->Blue);
			Image(i, j)->Red = (unsigned char)Temp;
			Image(i, j)->Green = (unsigned char)Temp;
			Image(i, j)->Blue = (unsigned char)Temp;
			imageAsArray[j * *cols + i] = Temp;
		}
	}
	return imageAsArray;
}

BMP readImageForCPU(char *filePathInput) {
	BMP Image;
	Image.ReadFromFile(filePathInput);
	// Преобразуем картику в черно-белую
	for (int i = 0; i < Image.TellWidth(); i++) {
		for (int j = 0; j < Image.TellHeight(); j++) {
			double Temp = 0.30*(Image(i, j)->Red) + 0.59*(Image(i, j)->Green) + 0.11*(Image(i, j)->Blue);
			Image(i, j)->Red = (unsigned char)Temp;
			Image(i, j)->Green = (unsigned char)Temp;
			Image(i, j)->Blue = (unsigned char)Temp;
		}
	}
	Image.SetBitDepth(8);
	CreateGrayscaleColorTable(Image);
	return Image;
}

void writeImage(char *filePath, float *grayscale, unsigned int rows, unsigned int cols) {
	BMP Output;
	Output.SetSize(cols, rows);
	// записали картинку 
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			RGBApixel pixel;
			pixel.Red = grayscale[i * cols + j];
			pixel.Green = grayscale[i * cols + j];
			pixel.Blue = grayscale[i * cols + j];
			pixel.Alpha = 0;
			Output.SetPixel(j, i, pixel);
		}
	}
	Output.WriteToFile(filePath);
}

BMP MedianFilterCPU(BMP image){
	float mask[COUNT_POINTS] = { 0,0,0,0,0,0,0,0,0 };
	BMP output;
	output.SetSize(image.TellHeight(), image.TellWidth());

	for (int row = 0; row < image.TellWidth(); row++) {
		for (int col = 0; col < image.TellHeight(); col++) {
			//Границы 0 заполнил
			if ((row == 0) || (col == 0) || (row == image.TellHeight() - 1) || (col == image.TellWidth() - 1)) { 
				RGBApixel pixel;
				pixel.Red = 0; pixel.Green = 0;	pixel.Blue = 0;
				output.SetPixel(col, row, pixel);
			}
			else {
				for (int x = 0; x < WINDOW_SIZE; x++) {
					for (int y = 0; y < WINDOW_SIZE; y++) {
						mask[x*WINDOW_SIZE + y] = image.GetPixel((col + y - 1), (row + x - 1)).Red; 
					}
				}

				// Отсортировали значения в маске 
				for (int i = 1; i < COUNT_POINTS; i++) {
					for (int j = i; j > 0 && mask[j - 1] > mask[j]; j--) {
						int tmp = mask[j - 1];
						mask[j - 1] = mask[j];
						mask[j] = tmp;
					}
				}
				RGBApixel pixel;
				pixel.Red = mask[4]; pixel.Green = mask[4];	pixel.Blue = mask[4];
				output.SetPixel(col, row, pixel);

			}
		}
	}
	return output;
}

int main() {
	setlocale(LC_ALL, "RUS");

	unsigned int rows, cols;
	// считали картинку 
	float * imageAsArray = readImage ("lena.bmp", &rows, &cols);
	std::cout << "Размер:" << rows<<"x"<<  cols<< std::endl;
	BMP imgCPU = readImageForCPU("lena.bmp");
	clock_t  start_time = clock();
	BMP outCPU = MedianFilterCPU(imgCPU);
	clock_t  end_time = clock();
	std::cout << "Время на CPU = " << (double)((end_time - start_time) * 1000 / CLOCKS_PER_SEC) << " мсек" << std::endl;
	outCPU.WriteToFile("resultCPU.bmp");


	//Создали дескриптор канала с форматом Float
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *cuArray;

	// Выделили Cuda массив соответствии со структурой hipChannelFormatDesc и вернули дескриптор нового массива CUDA в cuArray
	CUDA_CHECK_ERROR(hipMallocArray(&cuArray, &channelDesc, cols, rows));
	// Скопировали массив imageAsArray в cuArray
	CUDA_CHECK_ERROR(hipMemcpyToArray(cuArray, 0, 0, imageAsArray, rows * cols * sizeof(float), hipMemcpyHostToDevice));

	// Установили параметры текстуры
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModePoint;

	// Привязали массив к текстуре
	CUDA_CHECK_ERROR(hipBindTextureToArray(tex, cuArray, channelDesc));

	float *dev_output, *output;
	float gpuTime = 0;

	output = (float *)calloc(rows * cols, sizeof(float));

	CUDA_CHECK_ERROR(hipMalloc(&dev_output, rows * cols * sizeof(float)));

	dim3 dimBlock(16, 16);
	dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x,
		(rows + dimBlock.y - 1) / dimBlock.y);

	hipEvent_t start;
	hipEvent_t stop;

	//Создаем event'ы для синхронизации и замера времени работы GPU
	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	//Отмечаем старт расчетов на GPU
	hipEventRecord(start, 0);

	medianFilter << <dimGrid, dimBlock >> > (dev_output, cols, rows);

	//Копируем результат с девайса на хост в output
	CUDA_CHECK_ERROR(hipMemcpy(output, dev_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost));

	//Отмечаем окончание расчета
	hipEventRecord(stop, 0);

	//Синхронизируемя с моментом окончания расчетов
	hipEventSynchronize(stop);

	//Рассчитываем время работы GPU
	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout << "Время на GPU = " << gpuTime << " мсек" << std::endl;
	writeImage("result.bmp", output, rows, cols);

	//Чистим ресурсы на видеокарте
	CUDA_CHECK_ERROR(hipFreeArray(cuArray));
	CUDA_CHECK_ERROR(hipFree(dev_output));

	system("pause");
	return 0;
}
