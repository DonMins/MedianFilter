#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h> 
#include ""
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "bmp/EasyBMP.h"
#include <stdio.h>

using namespace std;

#define WINDOW_SIZE 3
#define COUNT_POINTS 9

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
system("pause");\
}       


// объявляем ссылку на текстуру для двумерной текстуры float
texture<float, hipTextureType2D, hipReadModeElementType> tex;


__global__ void medianFilter(float *output, int imageWidth, int imageHeight) {
	//  выбрали строку и столбец для потока
	int col = blockIdx.x *  blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	// маска по которой будет находиться медиана
	float mask[COUNT_POINTS] = { 0,0,0,0,0,0,0,0,0 };
	
	int k = 0;
	// Т.к текстуры обладают свойством свертывание - т.е выход за границы, будем идти от -1 до 1 с шагом 1 по картинки, 
	//заполняя маску  
	for (int i = -1; i <= 1; i++) {
		for (int j = -1; j <= 1; j++) {
			mask[k] = tex2D(tex, col + j, row + i);
			k++;
		}
	}

	// Отсортировали значения в маске 
	for (int i = 1; i < COUNT_POINTS; i++) {
		for (int j = i; j > 0 && mask[j - 1] > mask[j]; j--) {
			int tmp = mask[j - 1];
			mask[j - 1] = mask[j];
			mask[j] = tmp;
		}
	}
	// в результирующий центральный пиксель записали медиану
	output[row * imageWidth + col] = mask[4];
	
}


float *readImage(char *filePathInput, unsigned int *rows, unsigned int *cols) {
	BMP Image;
	Image.ReadFromFile(filePathInput);
	*rows = Image.TellHeight();
	*cols = Image.TellWidth();
	float *imageAsArray = (float *)calloc(*rows * *cols, sizeof(float));
	// Преобразуем картику в черно-белую
	for (int i = 0; i < Image.TellWidth(); i++)	{
		for (int j = 0; j < Image.TellHeight(); j++) {
			double Temp = 0.30*(Image(i, j)->Red) +	0.59*(Image(i, j)->Green) +	0.11*(Image(i, j)->Blue);
			Image(i, j)->Red = (unsigned char)Temp;
			Image(i, j)->Green = (unsigned char)Temp;
			Image(i, j)->Blue = (unsigned char)Temp;
			imageAsArray[j * *cols + i] = Temp;
		}
	}
	return imageAsArray;
}


void writeImage(char *filePath, float *grayscale, unsigned int rows, unsigned int cols) {
	BMP Output;
	Output.SetSize(cols, rows);
	// записали картинку 
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			RGBApixel pixel;
			pixel.Red = grayscale[i * cols + j];
			pixel.Green = grayscale[i * cols + j];
			pixel.Blue = grayscale[i * cols + j];
			pixel.Alpha = 0;
			Output.SetPixel(j, i, pixel);
		}
	}
	Output.WriteToFile(filePath);
}

int main() {
	setlocale(LC_ALL, "RUS");

	unsigned int rows, cols;
	// считали картинку 
	float * imageAsArray = readImage ("lena.bmp", &rows, &cols);

	//Создали дескриптор канала с форматом Float
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *cuArray;

	// Выделили Cuda массив соответствии со структурой hipChannelFormatDesc и вернули дескриптор нового массива CUDA в cuArray
	CUDA_CHECK_ERROR(hipMallocArray(&cuArray, &channelDesc, cols, rows));
	// Скопировали массив imageAsArray в cuArray
	CUDA_CHECK_ERROR(hipMemcpyToArray(cuArray, 0, 0, imageAsArray, rows * cols * sizeof(float), hipMemcpyHostToDevice));

	// Установили параметры текстуры
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModeLinear;

	// Привязали массив к текстуре
	CUDA_CHECK_ERROR(hipBindTextureToArray(tex, cuArray, channelDesc));

	float *dev_output, *output;
	float gpuTime = 0;

	output = (float *)calloc(rows * cols, sizeof(float));

	CUDA_CHECK_ERROR(hipMalloc(&dev_output, rows * cols * sizeof(float)));

	dim3 dimBlock(16, 16);
	dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x,
		(rows + dimBlock.y - 1) / dimBlock.y);

	hipEvent_t start;
	hipEvent_t stop;

	//Создаем event'ы для синхронизации и замера времени работы GPU
	CUDA_CHECK_ERROR(hipEventCreate(&start));
	CUDA_CHECK_ERROR(hipEventCreate(&stop));
	//Отмечаем старт расчетов на GPU
	hipEventRecord(start, 0);

	medianFilter << <dimGrid, dimBlock >> > (dev_output, cols, rows);

	//Копируем результат с девайса на хост в output
	CUDA_CHECK_ERROR(hipMemcpy(output, dev_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost));

	//Отмечаем окончание расчета
	hipEventRecord(stop, 0);

	//Синхронизируемя с моментом окончания расчетов
	hipEventSynchronize(stop);

	//Рассчитываем время работы GPU
	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout << "Время на GPU = " << gpuTime << " мсек" << std::endl;
	writeImage("result.bmp", output, rows, cols);

	//Чистим ресурсы на видеокарте
	CUDA_CHECK_ERROR(hipFreeArray(cuArray));
	CUDA_CHECK_ERROR(hipFree(dev_output));

	system("pause");
	return 0;
}
